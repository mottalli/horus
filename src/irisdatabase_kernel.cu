#include "hip/hip_runtime.h"
#include <vector>
#include <stdint.h>
#include <cassert>
#include <iostream>
#include <stdio.h>
#include "cudacommon.h"
#include "clock.h"

using namespace std;

// Taken from the CUDA SDK
#define CUDA_SAFE_CALL(err)           __cudaSafeCall      (err, __FILE__, __LINE__)
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
        fprintf(stderr, "CUDA_SAFE_CALL() Runtime API error in file <%s>, line %i : %s.\n",
                file, line, hipGetErrorString( err) );
        exit(-1);
    }
}


#define XOR(a, b, mask1, mask2) ((a ^ b) & mask1 & mask2)
#define MAX_ROTS 40
#define MAX_PARTS 8

__global__ void doGPUMatchKernel(const uint8_t* rotatedTemplates, const uint8_t* rotatedMasks, size_t nRotatedTemplates, const GPUDatabase database, float* distances)
{
	__shared__ float hammingDistances[MAX_ROTS];

	unsigned templateIdx = blockIdx.x;
	
	if (templateIdx > database.numberOfTemplates) {
		return;
	}
	
	size_t templateSize = database.templateWidth * database.templateHeight;
	size_t templateWords = templateSize / 4;			// 4 == sizeof(uint32_t);
	
	// Cast from chars to words
	uint32_t* rotatedTemplate = (uint32_t*)(rotatedTemplates + threadIdx.x*templateSize);
	uint32_t* rotatedMask = (uint32_t*)(rotatedMasks + threadIdx.x*templateSize);
	uint32_t* otherTemplate = (uint32_t*)(database.d_templates + templateIdx*templateSize);
	uint32_t* otherMask = (uint32_t*)(database.d_masks + templateIdx*templateSize);
	
	size_t nonZeroBits = 0, totalBits = 0;
	uint32_t word1, mask1;
	__shared__ uint32_t word2, mask2;
	
	for (size_t i = 0; i < templateWords; i++) {
		word1 = rotatedTemplate[i];
		mask1 = rotatedMask[i];
		if (threadIdx.x == 0) {
			word2 = otherTemplate[i];
			mask2 = otherMask[i];
		}
		__syncthreads();
		
		// __popc(x) returns the number of bits that are set to 1 in the binary representation of 32-bit integer parameter x.
		uint32_t x = XOR(word1, word2, mask1, mask2);
		nonZeroBits += __popc(x);
		totalBits += __popc(mask1 & mask2);
	}
	
	hammingDistances[threadIdx.x] = float(nonZeroBits) / float(totalBits);
	
	__syncthreads();
	
	if (threadIdx.x == 0) {
		float minHD = 1.0;
		for (int i = 0; i < blockDim.x; i++) {
			minHD = min(minHD, hammingDistances[i]);
		}
		distances[templateIdx] = minHD;
	}
}

__global__ void doGPUAContrarioMatchKernel(const uint8_t* rotatedTemplates, const uint8_t* rotatedMasks, size_t nRotatedTemplates, const GPUDatabase database, float* distances)
{
	__shared__ float hammingDistances[MAX_PARTS][MAX_ROTS];

	unsigned templateIdx = blockIdx.x;
	unsigned part = threadIdx.y;
	unsigned nParts = blockDim.y;

	//assert((database.templateWidth % 4) == 0);

	size_t templateSize = database.templateWidth * database.templateHeight;

	uint32_t* rotatedTemplate = (uint32_t*)(rotatedTemplates + threadIdx.x*templateSize);
	uint32_t* rotatedMask = (uint32_t*)(rotatedMasks + threadIdx.x*templateSize);
	uint32_t* otherTemplate = (uint32_t*)(database.d_templates + templateIdx*templateSize);
	uint32_t* otherMask = (uint32_t*)(database.d_masks + templateIdx*templateSize);

	unsigned widthRows = database.templateWidth / 4;		// Width of the template in 32-bit words
	unsigned partWidthWords = ceil(float(widthRows)/float(nParts));		// Width of the part in 32-bit words

	size_t nonZeroBits = 0, totalBits = 0;
	uint32_t word1, mask1;
	__shared__ uint32_t words2[MAX_ROTS], masks2[MAX_ROTS], word2, mask2;

	unsigned w0row = floor((float(widthRows)/float(nParts))*float(part));			// Offset of the first word in the part for each row

	unsigned idx;
	for (unsigned row = 0; row < database.templateHeight; row++) {
		for (unsigned col = 0; col < partWidthWords; col++) {
			idx = row*widthRows + w0row + col;
			word1 = rotatedTemplate[idx];
			mask1 = rotatedMask[idx];
			if (threadIdx.x == 0) {
				words2[part] = otherTemplate[idx];
				masks2[part] = otherMask[idx];
			}
			__syncthreads();

			word2 = words2[part];
			mask2 = masks2[part];

			uint32_t x = XOR(word1, word2, mask1, mask2);
			nonZeroBits += __popc(x);
			totalBits += __popc(mask1 & mask2);
		}
	}

	hammingDistances[part][threadIdx.x] = float(nonZeroBits) / float(totalBits);
	__syncthreads();

	if (threadIdx.x == 0) {
		float minHD = 1.0;
		for (int i = 0; i < blockDim.x; i++) {
			minHD = min(minHD, hammingDistances[part][i]);
		}
		distances[nParts*templateIdx+part] = minHD;
	}
}

/**
 * Load the database in the GPU
 */
void loadDatabase(const vector<IrisTemplate*>& templates, GPUDatabase& gpuDatabase)
{
	cleanupDatabase(&gpuDatabase);

	size_t n = templates.size();

	if (n <= 0) {
		return;
	}

	const Mat& sampleTemplate = templates[0]->getPackedTemplate();
	size_t templateWidth = sampleTemplate.cols, templateHeight = sampleTemplate.rows;
	size_t templateSize = templateWidth*templateHeight;
	gpuDatabase.templateWidth = templateWidth;
	gpuDatabase.templateHeight = templateHeight;
	gpuDatabase.numberOfTemplates = n;

	size_t bytes = n*templateSize;
	CUDA_SAFE_CALL(hipMalloc(&gpuDatabase.d_templates, bytes));
	CUDA_SAFE_CALL(hipMalloc(&gpuDatabase.d_masks, bytes));

	for (size_t i = 0; i < n; i++) {
		const Mat& packedTemplate = templates[i]->getPackedTemplate();
		const Mat& packedMask = templates[i]->getPackedMask();

		assert(packedTemplate.isContinuous() && packedMask.isContinuous());
		assert(packedTemplate.channels() == 1 && packedMask.channels() == 1);
		assert(packedTemplate.type() == CV_8U && packedMask.type() == CV_8U);
		assert(packedTemplate.size() == packedMask.size());
		assert(packedTemplate.cols == gpuDatabase.templateWidth);
		assert(packedTemplate.rows == gpuDatabase.templateHeight);

		// Copy the template and mask to the GPU
		CUDA_SAFE_CALL(hipMemcpy(gpuDatabase.d_templates + i*templateSize, packedTemplate.data, templateSize, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(gpuDatabase.d_masks + i*templateSize, packedMask.data, templateSize, hipMemcpyHostToDevice));
	}
};

void doGPUMatch(const TemplateComparator& comparator, GPUDatabase& gpuDatabase, vector<double>& resultDistances, double& matchingTime)
{
	Clock clock;
	clock.start();

	const std::vector<IrisTemplate>& rotatedTemplates = comparator.rotatedTemplates;
	size_t n = gpuDatabase.numberOfTemplates;

	assert(rotatedTemplates.size() < MAX_ROTS);

	// Load the rotated templates and masks to the GPU
	size_t templateSize = gpuDatabase.templateWidth * gpuDatabase.templateHeight;
	uint8_t *d_rotatedTemplates, *d_rotatedMasks;
	size_t bytes = rotatedTemplates.size() * templateSize;

	CUDA_SAFE_CALL(hipMalloc(&d_rotatedTemplates, bytes));
	CUDA_SAFE_CALL(hipMalloc(&d_rotatedMasks, bytes));
	for (size_t i = 0; i < rotatedTemplates.size(); i++) {
		const Mat& packedTemplate = rotatedTemplates[i].getPackedTemplate();
		const Mat& packedMask = rotatedTemplates[i].getPackedMask();

		assert(packedTemplate.isContinuous() && packedMask.isContinuous());
		assert(packedTemplate.channels() == 1 && packedMask.channels() == 1);
		assert(packedTemplate.type() == CV_8U && packedMask.type() == CV_8U);
		assert(packedTemplate.size() == packedMask.size());
		assert(packedTemplate.cols == gpuDatabase.templateWidth);
		assert(packedTemplate.rows == gpuDatabase.templateHeight);

		CUDA_SAFE_CALL(hipMemcpy(d_rotatedTemplates + i*templateSize, packedTemplate.data, templateSize, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_rotatedMasks + i*templateSize, packedMask.data, templateSize, hipMemcpyHostToDevice));
	}

	// Output buffer in device
	float* d_distances;
	CUDA_SAFE_CALL(hipMalloc(&d_distances, n*sizeof(float)));


	// Invoke the kernel
	dim3 blockSize(rotatedTemplates.size(), 1, 1);
	dim3 gridSize(n, 1);

	doGPUMatchKernel<<<gridSize, blockSize>>>(
		d_rotatedTemplates,
		d_rotatedMasks,
		rotatedTemplates.size(),
		gpuDatabase,
		d_distances
	);

	// Retrieve the result
	float* distances = new float[n];
	hipMemcpy(distances, d_distances, n*sizeof(float), hipMemcpyDeviceToHost);


	// Copy the results (cast to double)
	for (size_t i = 0; i < n; i++) {
		resultDistances[i] = double(distances[i]);
	}

	// Free the memory
	CUDA_SAFE_CALL(hipFree(d_rotatedTemplates));
	CUDA_SAFE_CALL(hipFree(d_rotatedMasks));
	CUDA_SAFE_CALL(hipFree(d_distances));
	free(distances);

	matchingTime = clock.stop();
};

void doGPUAContrarioMatch(const TemplateComparator& comparator, GPUDatabase& gpuDatabase, unsigned nParts, vector< vector<double> >& resultDistances, double& matchingTime)
{
	assert(resultDistances.size() == nParts);

	const std::vector<IrisTemplate>& rotatedTemplates = comparator.rotatedTemplates;
	size_t n = gpuDatabase.numberOfTemplates;

	assert(rotatedTemplates.size() < MAX_ROTS);

	Clock clock;
	clock.start();

	// Load the rotated templates and masks to the GPU
	size_t templateSize = gpuDatabase.templateWidth * gpuDatabase.templateHeight;
	uint8_t *d_rotatedTemplates, *d_rotatedMasks;
	size_t bytes = rotatedTemplates.size() * templateSize;

	CUDA_SAFE_CALL(hipMalloc(&d_rotatedTemplates, bytes));
	CUDA_SAFE_CALL(hipMalloc(&d_rotatedMasks, bytes));
	for (size_t i = 0; i < rotatedTemplates.size(); i++) {
		const Mat& packedTemplate = rotatedTemplates[i].getPackedTemplate();
		const Mat& packedMask = rotatedTemplates[i].getPackedMask();

		assert(packedTemplate.isContinuous() && packedMask.isContinuous());
		assert(packedTemplate.channels() == 1 && packedMask.channels() == 1);
		assert(packedTemplate.type() == CV_8U && packedMask.type() == CV_8U);
		assert(packedTemplate.size() == packedMask.size());
		assert(packedTemplate.cols == gpuDatabase.templateWidth);
		assert(packedTemplate.rows == gpuDatabase.templateHeight);

		CUDA_SAFE_CALL(hipMemcpy(d_rotatedTemplates + i*templateSize, packedTemplate.data, templateSize, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_rotatedMasks + i*templateSize, packedMask.data, templateSize, hipMemcpyHostToDevice));
	}


	// Output buffer in device
	float* d_distances;
	CUDA_SAFE_CALL(hipMalloc(&d_distances, n*nParts*sizeof(float)));


	// Invoke the kernel
	dim3 blockSize(rotatedTemplates.size(), nParts, 1);
	dim3 gridSize(n, 1);

	doGPUAContrarioMatchKernel<<<gridSize, blockSize>>>(
		d_rotatedTemplates,
		d_rotatedMasks,
		rotatedTemplates.size(),
		gpuDatabase,
		d_distances
	);

	// Retrieve the result
	float* distances = new float[n*nParts];
	hipMemcpy(distances, d_distances, n*sizeof(float)*nParts, hipMemcpyDeviceToHost);


	// Copy the results
	for (size_t i = 0; i < n; i++) {
		for (size_t p = 0; p < nParts; p++) {
			resultDistances[p][i] = double(distances[i*nParts+p]);
		}
	}

	// Free the memory
	CUDA_SAFE_CALL(hipFree(d_rotatedTemplates));
	CUDA_SAFE_CALL(hipFree(d_rotatedMasks));
	CUDA_SAFE_CALL(hipFree(d_distances));
	free(distances);

	matchingTime = clock.stop();
};

void cleanupDatabase(GPUDatabase* database)
{
	if (database->d_templates != NULL) {
		hipFree(database->d_templates);
		hipFree(database->d_masks);
	}
}
