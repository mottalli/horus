#include "hip/hip_runtime.h"
#include <vector>
#include <stdint.h>
#include <cassert>
#include <iostream>
#include "cudacommon.h"
#include "clock.h"

using namespace std;

// Taken from the CUDA SDK
#define CUDA_SAFE_CALL(err)           __cudaSafeCall      (err, __FILE__, __LINE__)
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
        fprintf(stderr, "CUDA_SAFE_CALL() Runtime API error in file <%s>, line %i : %s.\n",
                file, line, hipGetErrorString( err) );
        exit(-1);
    }
}


#define XOR(a, b, mask1, mask2) ((a ^ b) & mask1 & mask2)
#define MAX_ROTS 40
#define MAX_PARTS 8

__global__ void doGPUMatchKernel(const uint8_t* rotatedTemplates, const uint8_t* rotatedMasks, size_t nRotatedTemplates, const GPUDatabase database, float* distances)
{
	__shared__ float hammingDistances[MAX_ROTS];

	unsigned templateIdx = blockIdx.x;
	
	if (templateIdx > database.numberOfTemplates) {
		return;
	}
	
	size_t templateSize = database.templateWidth * database.templateHeight;
	size_t templateWords = templateSize / 4;			// 4 == sizeof(uint32_t);
	
	// Cast from chars to words
	uint32_t* rotatedTemplate = (uint32_t*)(rotatedTemplates + threadIdx.x*templateSize);
	uint32_t* rotatedMask = (uint32_t*)(rotatedMasks + threadIdx.x*templateSize);
	uint32_t* otherTemplate = (uint32_t*)(database.d_templates + templateIdx*templateSize);
	uint32_t* otherMask = (uint32_t*)(database.d_masks + templateIdx*templateSize);
	
	size_t nonZeroBits = 0, totalBits = 0;
	uint32_t word1, mask1;
	__shared__ uint32_t word2, mask2;
	
	for (size_t i = 0; i < templateWords; i++) {
		word1 = rotatedTemplate[i];
		mask1 = rotatedMask[i];
		if (threadIdx.x == 0) {
			word2 = otherTemplate[i];
			mask2 = otherMask[i];
		}
		__syncthreads();
		
		// __popc(x) returns the number of bits that are set to 1 in the binary representation of 32-bit integer parameter x.
		uint32_t x = XOR(word1, word2, mask1, mask2);
		nonZeroBits += __popc(x);
		totalBits += __popc(mask1 & mask2);
	}
	
	hammingDistances[threadIdx.x] = float(nonZeroBits) / float(totalBits);
	
	__syncthreads();
	
	if (threadIdx.x == 0) {
		float minHD = 1.0;
		for (int i = 0; i < blockDim.x; i++) {
			minHD = min(minHD, hammingDistances[i]);
		}
		distances[templateIdx] = minHD;
	}
}

__global__ void doGPUAContrarioMatchKernel(const uint8_t* rotatedTemplates, const uint8_t* rotatedMasks, size_t nRotatedTemplates, const GPUDatabase database, float* distances)
{
	__shared__ float hammingDistances[MAX_PARTS][MAX_ROTS];

	unsigned templateIdx = blockIdx.x;
	unsigned part = threadIdx.y;
	unsigned nParts = blockDim.y;

	assert((database.templateWidth % 4) == 0);

	size_t templateSize = database.templateWidth * database.templateHeight;

	uint32_t* rotatedTemplate = (uint32_t*)(rotatedTemplates + threadIdx.x*templateSize);
	uint32_t* rotatedMask = (uint32_t*)(rotatedMasks + threadIdx.x*templateSize);
	uint32_t* otherTemplate = (uint32_t*)(database.d_templates + templateIdx*templateSize);
	uint32_t* otherMask = (uint32_t*)(database.d_masks + templateIdx*templateSize);

	unsigned widthRows = database.templateWidth / 4;		// Width of the template in 32-bit words
	unsigned partWidthWords = ceil(float(widthRows)/float(nParts));		// Width of the part in 32-bit words

	size_t nonZeroBits = 0, totalBits = 0;
	uint32_t word1, mask1;
	__shared__ uint32_t words2[MAX_ROTS], masks2[MAX_ROTS], word2, mask2;

	unsigned w0row = floor((float(widthRows)/float(nParts))*float(part));			// Offset of the first word in the part for each row

	unsigned idx;
	for (unsigned row = 0; row < database.templateHeight; row++) {
		for (unsigned col = 0; col < partWidthWords; col++) {
			idx = row*widthRows + w0row + col;
			word1 = rotatedTemplate[idx];
			mask1 = rotatedMask[idx];
			if (threadIdx.x == 0) {
				words2[part] = otherTemplate[idx];
				masks2[part] = otherMask[idx];
			}
			__syncthreads();

			word2 = words2[part];
			mask2 = masks2[part];

			uint32_t x = XOR(word1, word2, mask1, mask2);
			nonZeroBits += __popc(x);
			totalBits += __popc(mask1 & mask2);
		}
	}

	hammingDistances[part][threadIdx.x] = float(nonZeroBits) / float(totalBits);
	__syncthreads();

	if (threadIdx.x == 0) {
		float minHD = 1.0;
		for (int i = 0; i < blockDim.x; i++) {
			minHD = min(minHD, hammingDistances[part][i]);
		}
		distances[nParts*templateIdx+part] = minHD;
	}
}

/**
 * Load the database in the GPU
 */
void loadDatabase(const vector<const uint8_t*>& templates, const vector<const uint8_t*>& masks, size_t templateWidth, size_t templateHeight, GPUDatabase* database)
{
	assert(templateWidth % 4 == 0);			// For casting to int32 in the GPU (4x speedup)
	size_t templateSize = templateWidth*templateHeight;

	cleanupDatabase(database);

	database->templateWidth = templateWidth;
	database->templateHeight = templateHeight;
	database->numberOfTemplates = templates.size();

	size_t bytes = templates.size()*templateSize;
	CUDA_SAFE_CALL(hipMalloc(&database->d_templates, bytes));
	CUDA_SAFE_CALL(hipMalloc(&database->d_masks, bytes));

	// Load each individual template in a contiguous chunk of GPU memory
	for (size_t i = 0; i < templates.size(); i++) {
		CUDA_SAFE_CALL(hipMemcpy(database->d_templates + i*templateSize, templates[i], templateSize, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(database->d_masks + i*templateSize, masks[i], templateSize, hipMemcpyHostToDevice));
	}
};

void doGPUMatch(const vector<const uint8_t*>& rotatedTemplates, const vector<const uint8_t*>& rotatedMasks, GPUDatabase* database, vector<double>& resultDistances, double& matchingTime)
{
	assert(rotatedTemplates.size() == rotatedMasks.size());
	assert(rotatedTemplates.size() < MAX_ROTS);
	assert(resultDistances.size() == database->numberOfTemplates);

	Clock clock;
	clock.start();

	// Load the rotated templates and masks to the GPU
	uint8_t *d_rotatedTemplates, *d_rotatedMasks;
	size_t templateSize = database->templateWidth * database->templateHeight;
	size_t bytes = rotatedTemplates.size() * templateSize;
	
	CUDA_SAFE_CALL(hipMalloc(&d_rotatedTemplates, bytes));
	CUDA_SAFE_CALL(hipMalloc(&d_rotatedMasks, bytes));
	for (size_t i = 0; i < rotatedTemplates.size(); i++) {
		CUDA_SAFE_CALL(hipMemcpy(d_rotatedTemplates + i*templateSize, rotatedTemplates[i], templateSize, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_rotatedMasks + i*templateSize, rotatedMasks[i], templateSize, hipMemcpyHostToDevice));
	}

	// Output buffer in device
	float* d_distances;
	CUDA_SAFE_CALL(hipMalloc(&d_distances, database->numberOfTemplates*sizeof(float)));


	// Invoke the kernel
	dim3 blockSize(rotatedTemplates.size(), 1, 1);
	dim3 gridSize(database->numberOfTemplates, 1);

	doGPUMatchKernel<<<gridSize, blockSize>>>(
		d_rotatedTemplates,
		d_rotatedMasks,
		rotatedTemplates.size(),
		*database,
		d_distances
	);

	// Retrieve the result
	float* distances = new float[database->numberOfTemplates];
	hipMemcpy(distances, d_distances, database->numberOfTemplates*sizeof(float), hipMemcpyDeviceToHost);


	// Copy the results
	for (size_t i = 0; i < database->numberOfTemplates; i++) {
		resultDistances[i] = double(distances[i]);
	}

	// Free the memory
	CUDA_SAFE_CALL(hipFree(d_rotatedTemplates));
	CUDA_SAFE_CALL(hipFree(d_rotatedMasks));
	CUDA_SAFE_CALL(hipFree(d_distances));
	free(distances);

	matchingTime = clock.stop();
};

void doGPUAContrarioMatch(const vector<const uint8_t*>& rotatedTemplates, const vector<const uint8_t*>& rotatedMasks, GPUDatabase* database, unsigned nParts, vector< vector<double> >& resultDistances, double& matchingTime)
{
	assert(rotatedTemplates.size() == rotatedMasks.size());
	assert(rotatedTemplates.size() < MAX_ROTS);
	assert(resultDistances.size() == database->numberOfTemplates);

	Clock clock;
	clock.start();

	// Load the rotated templates and masks to the GPU
	uint8_t *d_rotatedTemplates, *d_rotatedMasks;
	size_t templateSize = database->templateWidth * database->templateHeight;
	size_t bytes = rotatedTemplates.size() * templateSize;

	CUDA_SAFE_CALL(hipMalloc(&d_rotatedTemplates, bytes));
	CUDA_SAFE_CALL(hipMalloc(&d_rotatedMasks, bytes));
	for (size_t i = 0; i < rotatedTemplates.size(); i++) {
		CUDA_SAFE_CALL(hipMemcpy(d_rotatedTemplates + i*templateSize, rotatedTemplates[i], templateSize, hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipMemcpy(d_rotatedMasks + i*templateSize, rotatedMasks[i], templateSize, hipMemcpyHostToDevice));
	}

	// Output buffer in device
	float* d_distances;
	CUDA_SAFE_CALL(hipMalloc(&d_distances, database->numberOfTemplates*nParts*sizeof(float)));


	// Invoke the kernel
	dim3 blockSize(rotatedTemplates.size(), nParts, 1);
	dim3 gridSize(database->numberOfTemplates, 1);

	doGPUAContrarioMatchKernel<<<gridSize, blockSize>>>(
		d_rotatedTemplates,
		d_rotatedMasks,
		rotatedTemplates.size(),
		*database,
		d_distances
	);

	// Retrieve the result
	float* distances = new float[database->numberOfTemplates*nParts];
	hipMemcpy(distances, d_distances, database->numberOfTemplates*sizeof(float)*nParts, hipMemcpyDeviceToHost);


	// Copy the results
	for (size_t i = 0; i < database->numberOfTemplates; i++) {
		for (size_t p = 0; p < nParts; p++) {
			resultDistances[p][i] = double(distances[i*nParts+p]);
		}
	}

	// Free the memory
	CUDA_SAFE_CALL(hipFree(d_rotatedTemplates));
	CUDA_SAFE_CALL(hipFree(d_rotatedMasks));
	CUDA_SAFE_CALL(hipFree(d_distances));
	free(distances);

	matchingTime = clock.stop();
};

void cleanupDatabase(GPUDatabase* database)
{
	if (database->d_templates != NULL) {
		hipFree(database->d_templates);
		hipFree(database->d_masks);
	}
}
